#include "hip/hip_runtime.h"
﻿#include <cstdio>
#include <cstdlib>
#include <ctime>
#include <iostream>
#include <fstream>
#include <iomanip>
#include <chrono>

struct Data {
    int N, R, BS, K;
};

Data data[] = {
    //nasycenie N
       // {1536, 32, 32, 8},
       // {1536, 32, 32, 8},
      //  {1536, 16, 32, 8},
      //  {1536, 16, 32, 4},
       // {1536, 8, 32, 4},
       // {1536, 8, 32, 4},

      //  {832,  32, 32, 8},
       // {832,  32, 32, 8},
       // {832,  16, 32, 8},
       // {832,  16, 32, 4},
       // {832,  8, 32, 4},
       // {832,  8, 32, 4},

      //  {512,  32, 32, 8},
       // {512,  32, 32, 8},
       // {512,  16, 32, 8},
       // {512,  16, 32, 4},
       // {512,  8, 32, 4},
       // {512,  8, 32, 4},

       // {256, 32, 32, 8},
       // {256, 32, 32, 8},
       // {256, 16, 32, 8},
        //{256, 16, 32, 4},
       // {256, 8, 32, 4},
       // {256, 8, 32, 4},

        //{164, 32, 32, 8},
        //{164, 32, 32, 8},
       // {164, 16, 32, 8},
       // {164, 16, 32, 4},
       // {164, 8, 32, 4},
       // {164, 8, 32, 4},

       // {80,  32, 32, 8},
       // {80,  32, 32, 8},
       // {80,  16, 32, 8},
       // {80,  16, 32, 4},
       // {80,  8, 32, 4},
       // {80,  8, 32, 4},

        //nasycenie K
             {832,  32, 16, 32},
             {832,  32, 16, 28},
             {832,  32, 16, 24},
             {832,  32, 16, 22},
             {832,  32, 16, 20},
             {832,  32, 16, 18},
             {832,  32, 16, 16},
             {832,  32, 16, 12},
             {832,  32, 16, 10},
             {832,  32, 16, 8},
             {832,  32, 16, 6},
             {832,  32, 16, 4},
};


void calculateMatrixSeq(int N, int R, float* mat, float* out) {
    for (int i = R; i < N - R; i++) {
        for (int j = R; j < N - R; j++) {
            float sum = 0;
            for (int x = i - R; x <= i + R; x++) {
                for (int y = j - R; y <= j + R; y++) {
                    sum += mat[x * N + y];
                }
            }
            out[(i - R) * (N - 2 * R) + j - R] = sum;
        }
    }
}

void compareMatrices(const float* out1, const float* out2, const int N, const int R) {
    int out_size = N - 2 * R;

    for (int i = 0; i < out_size * out_size - 1; i++) {
        if ((out1[i] != out2[i])) {
            fprintf(stderr, "Error in [%d] %f != %f.\n", i, out2[i], out1[i]);
            exit(1);
        }
    }
}

__global__ void calculateMatrixGlobal(const float* mat, float* out, const int N, const int R, const int K)
{
    int out_size = N - 2 * R;
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = (threadIdx.y + blockIdx.y * blockDim.y) * K;

    for (int k = 0; k < K; k++)
    {
        float sum = 0;
        if (i < out_size && j + k < out_size) {
            for (int ry = -R; ry <= R; ry++) {
                for (int rx = -R; rx <= R; rx++) {
                    sum += mat[(j + k + R + ry) * N + (i + R + rx)];
                }
            }
            out[(j + k) * out_size + i] = sum;
        }
    }
}

__device__ inline int translateIndex(int index, int neededWidth, int xOffset, int yOffset, int N) {
    int mat_x = index / neededWidth;
    int mat_y = index % neededWidth;
    return (mat_y + xOffset) * N + (mat_x + yOffset);
}

__global__ void calculateMatrixShared(const float* mat, float* out, int N, int R, int K) {
    extern __shared__ float ref[];

    unsigned int i = ((blockIdx.x * blockDim.x) + threadIdx.x) + R;
    unsigned int j = ((blockIdx.y * blockDim.y * K) + threadIdx.y) + R;

    unsigned int neededWidth = N - 2 * R - blockIdx.x * blockDim.x >= blockDim.x ? blockDim.x + 2 * R : N - 2 * R - blockIdx.x * blockDim.x + 2 * R;
    if (N < (blockDim.x + 2 * R)) neededWidth = N;

    unsigned int neededHeight = N - 2 * R - blockIdx.y * blockDim.x >= blockDim.x ? blockDim.x + 2 * R : N - 2 * R - blockIdx.y * blockDim.x + 2 * R;
    if (N < (blockDim.x + 2 * R)) neededHeight = N;

    unsigned int neededSize = neededHeight * neededWidth;
    unsigned int threadNum = threadIdx.x * blockDim.x + threadIdx.y;
    for (int k = 0; k < K; k++) {
        for (unsigned int ii = threadNum; ii < neededSize; ii += blockDim.x * blockDim.x) {
            int threadInd = translateIndex(ii, neededWidth, blockIdx.x * blockDim.x, blockIdx.y * blockDim.y * K + k * blockDim.x, N);
            ref[ii] = mat[threadInd];
        }
        __syncthreads();

        unsigned int threadx = threadIdx.x;
        unsigned int thready = threadIdx.y;

        if (i < N - R && j < N - R) {
            float sum = 0;
            for (unsigned int x = threadx; x <= threadx + 2 * R; x++)
                for (unsigned int y = thready; y <= thready + 2 * R; y++)
                    sum += ref[y * neededWidth + x];

            out[(i - R) * (N - 2 * R) + (j - R)] = sum;
            j += blockDim.x;
        }
        __syncthreads();
    }
}

void randomizeArray(float* mat, int N) {
    srand(time(NULL));
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            mat[i * N + j] = (int)(rand() % 100);
        }
    }
}

void calculateMatrix(const int N, const int R, const int K, const int OUT_SIZE, const char* name, const float* mat, const float* out_seq, const dim3 threadsMatrix, const dim3 blocksMatrix, const int sharedMemSize) {
    float* copy_mat, * copy_out;
    hipEvent_t start, stop;
    float time = 0;

    auto* out = (float*)malloc(OUT_SIZE * OUT_SIZE * sizeof(float));

    hipMalloc((void**)&copy_mat, N * N * sizeof(float));
    hipMalloc((void**)&copy_out, OUT_SIZE * OUT_SIZE * sizeof(float));
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipMemcpyAsync(copy_mat, mat, N * N * sizeof(float), hipMemcpyHostToDevice);
    hipEventRecord(start, nullptr);

    if (strcmp(name, "global") == 0) {
        calculateMatrixGlobal << <blocksMatrix, threadsMatrix >> > (copy_mat, copy_out, N, R, K);
    }
    else {
        calculateMatrixShared << <blocksMatrix, threadsMatrix, sharedMemSize >> > (copy_mat, copy_out, N, R, K);
    }

    hipEventRecord(stop, nullptr);
    hipMemcpyAsync(out, copy_out, OUT_SIZE * OUT_SIZE * sizeof(float), hipMemcpyDeviceToHost);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(copy_mat);
    hipFree(copy_out);
    compareMatrices(out_seq, out, N, R);
    free(out);

    printf("%f;", time);
}

int main() {
    for (auto& one : data) {
        int N = one.N;
        int R = one.R;
        int BS = one.BS;
        int K = one.K;
        int OUT_SIZE = one.N - one.R * 2;
        printf("%d;%d;%d;%d;", N, R, BS, K);

        auto* out = (float*)malloc(OUT_SIZE * OUT_SIZE * sizeof(float));

        dim3 threadsMatrix(BS, BS);
        dim3 blocksMatrix(ceil(OUT_SIZE / (float)BS), ceil(OUT_SIZE / (float)BS / K));
        int sharedMemSize = sizeof(float) * (BS + 2 * R) * (BS + 2 * R);

        auto* mat = (float*)malloc(N * N * sizeof(float));
        randomizeArray(mat, N);

        auto startSeq = std::chrono::high_resolution_clock::now();
        calculateMatrixSeq(N, R, (float*)mat, (float*)out);
        auto stopSeq = std::chrono::high_resolution_clock::now();
        auto timeSeq = std::chrono::duration_cast<std::chrono::microseconds>(stopSeq - startSeq);
        printf("%f;", timeSeq.count() / 1000.0f);

        calculateMatrix(N, R, K, OUT_SIZE, "global", mat, out, threadsMatrix, blocksMatrix, sharedMemSize);
        calculateMatrix(N, R, K, OUT_SIZE, "shared", mat, out, threadsMatrix, blocksMatrix, sharedMemSize);

        printf("\n");

        free(mat);
        free(out);
    }
    return 0;
}
